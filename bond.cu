
#include <hip/hip_runtime.h>
#include<iostream>
#include <cassert>  

#define KERNEL extern "C" __global__

inline __device__ float3 make_float3(float a) {
    return make_float3(a, a, a);
}

inline __device__ void operator*=(float3& a, float b) {
    a.x *= b; a.y *= b; a.z *= b;
}

inline __device__ void operator*=(float3& a, float3 b) {
    a.x *= b.x; a.y *= b.y; a.z *= b.z;
}

inline __device__ void operator+=(float3& a, float3 b) {
    a.x += b.x; a.y += b.y; a.z += b.z;
}

inline __device__ float3 operator-(float3 a) {
    return make_float3(-a.x, -a.y, -a.z);
}

__device__ inline long long realToFixedPoint(float x) {
    return static_cast<long long>(x * 0x100000000);
}

extern "C" __global__ void computeBondedForces(unsigned long long* __restrict__ forceBuffer,
 float* __restrict__ energyBuffer, const float4* __restrict__ posq,
  const uint2* __restrict__ atomIndices0_0, float2* params, int num_atoms) {
float energy = 0;
for (unsigned int index = blockIdx.x*blockDim.x+threadIdx.x; index < num_atoms-1; index += blockDim.x*gridDim.x) {
    uint2 atoms0 = atomIndices0_0[index];
    unsigned int atom1 = atoms0.x;
    float4 pos1 = posq[atom1];
    unsigned int atom2 = atoms0.y;
    float4 pos2 = posq[atom2];
float3 delta = make_float3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
#if 0
APPLY_PERIODIC_TO_DELTA(delta)
#endif
float r = sqrtf(delta.x*delta.x + delta.y*delta.y + delta.z*delta.z);
float2 bondParams = params[index];
float deltaIdeal = r-bondParams.x;
energy += 0.5f * bondParams.y*deltaIdeal*deltaIdeal;
float dEdR = bondParams.y * deltaIdeal;

dEdR = (r > 0) ? (dEdR / r) : 0;
delta *= dEdR;
float3 force1 = delta;
float3 force2 = -delta;

    atomicAdd(&forceBuffer[atom1], static_cast<unsigned long long>(realToFixedPoint(force1.x)));
    atomicAdd(&forceBuffer[atom1+num_atoms], static_cast<unsigned long long>(realToFixedPoint(force1.y)));
    atomicAdd(&forceBuffer[atom1+num_atoms*2], static_cast<unsigned long long>(realToFixedPoint(force1.z)));
    __threadfence_block();
    atomicAdd(&forceBuffer[atom2], static_cast<unsigned long long>(realToFixedPoint(force2.x)));
    atomicAdd(&forceBuffer[atom2+num_atoms], static_cast<unsigned long long>(realToFixedPoint(force2.y)));
    atomicAdd(&forceBuffer[atom2+num_atoms*2], static_cast<unsigned long long>(realToFixedPoint(force2.z)));
    __threadfence_block();
}
energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
//printf("%6.4lf\n", energy);
}


int main(int argc, char** argv) {

if (argc != 2) {
    printf("Usage: %s <NINT>\n", argv[0]);
    exit(1);
}

int num_atoms = (atoi(argv[1])*200)+1;
int num_bonds = num_atoms-1;

//x, y, z for atoms which have a spring connected to 0,0,0
float p = 5.0;
//spring constant for all springs
float k = 2;
//equilibrium separation for all springs
float r0 = 1;

//make bond idxs (these ids denote what atoms have springs between them)
uint2* atomIndices0_0 = new uint2[num_bonds]();
uint2* atomIndices0_0_h = new uint2[num_bonds]();
for (unsigned int i = 0; i < num_bonds; i++) {
  atomIndices0_0_h[i] = {0, i+1};
}

hipMalloc((void **) &atomIndices0_0, num_bonds*sizeof(uint2));
hipMemcpy(atomIndices0_0, atomIndices0_0_h, num_bonds*sizeof(uint2), hipMemcpyHostToDevice);

//make bond params, (r0, k)
float2* params = new float2[num_bonds]();
float2* params_h = new float2[num_bonds]();
for (int i = 0; i < num_bonds; i++) {
  params_h[i] = {r0, k};
}

hipMalloc((void **) &params, num_bonds*sizeof(float2));
hipMemcpy(params, params_h, num_bonds*sizeof(float2), hipMemcpyHostToDevice);

//make energy running total
float* energyBuffer = new float[num_bonds]();
float* energyBuffer_h = new float[num_bonds]();
for (int i = 0; i < num_bonds; i++) {
  energyBuffer_h[i] = 0.0;
}

hipMalloc((void **) &energyBuffer, num_bonds*sizeof(float));
hipMemcpy(energyBuffer, energyBuffer_h, num_bonds*sizeof(float), hipMemcpyHostToDevice);

//make force running total
unsigned long long* forceBuffer = new unsigned long long[num_atoms*3]();
unsigned long long* forceBuffer_h = new unsigned long long[num_atoms*3]();
for (int i = 0; i < num_atoms*3; i++) {
  forceBuffer_h[i] = 0.0;
}

hipMalloc((void **) &forceBuffer, 3*num_atoms*sizeof(unsigned long long));
hipMemcpy(forceBuffer, forceBuffer_h, 3*num_atoms*sizeof(unsigned long long), hipMemcpyHostToDevice);

//make positions (Place one atom at 0,0,0 and the rest at some user defined x,y,z)
float4* posq = new float4[num_atoms]();
float4* posq_h = new float4[num_atoms]();
posq_h[0] = {0.0,0.0,0.0,0.0};
for (int i = 1; i < num_atoms; i++) {
  posq_h[i] = {p,p,p,0.0};
}

hipMalloc((void **) &posq, num_atoms*sizeof(float4));
hipMemcpy(posq, posq_h, num_atoms*sizeof(float4), hipMemcpyHostToDevice);

computeBondedForces<<<num_bonds/200, 200>>>(forceBuffer, energyBuffer, posq, atomIndices0_0, params, num_atoms);


hipMemcpy(energyBuffer_h, energyBuffer, num_bonds*sizeof(float), hipMemcpyDeviceToHost);
for (int i = 0; i < num_bonds; i++) {
    //0.5*k*(sqrt(3*p**2)-r0)**2
    //0.5*2*(sqrt(75)-1)**2
    assert(abs(energyBuffer_h[i]-58.6795) <= 0.0001);
    //printf("%6.4lf\n", energyBuffer_h[i]); 
}

hipError_t cudaerr = hipDeviceSynchronize();
if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
        hipGetErrorString(cudaerr));
}

